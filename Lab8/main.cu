

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

#define TILE_WIDTH 16

__global__ void sum_matrices(float *ma, float *mb, float *mc, int height, int width)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y; // linie
    int col = blockIdx.x * blockDim.x + threadIdx.x; // coloana
    
    // se poate si invers, adica row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
	    mc[row * height + col] = ma[row * height + col] + mb[row * height + col];
    }
}

int main() {
	// size
    const size_t n = 1 << 6;
    
    // setam dimensiunea unui bloc pentru linie, respectiv coloana
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    
    // determinam numarul de blocuri pentru linie, respectiv coloana
    const dim3 num_blocks(n / block_size.x, n / block_size.y);
    // nu avem dim2 in CUDA

    // alocam memorie pentru host
    float *host_a = 0, *host_b = 0, *host_c = 0;
	host_a = (float *) malloc(n * n * sizeof(float));
    host_b = (float *) malloc(n * n * sizeof(float));
    host_c = (float *) malloc(n * n * sizeof(float));
    
	for (int i = 0; i < n * n; i++) {
		host_a[i] = 2;
        host_b[i] = 4;
        host_c[i] = 0;
	}

    // alocam memorie pentru device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void**)&device_a, sizeof(float) * n * n);
	hipMalloc((void**)&device_b, sizeof(float) * n * n);
	hipMalloc((void**)&device_c, sizeof(float) * n * n);

    // transfer date CPU -> GPU
	hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(device_c, &host_c[0], sizeof(float) * n * n, hipMemcpyHostToDevice);

    // evenimente CUDA, pe care le folosim pentru masurarea timpului de executie
    hipEvent_t launch_begin, launch_end;

    // creeam evenimentele
	hipEventCreate(&launch_begin);
	hipEventCreate(&launch_end);
    
    // lansam in executie evenimentul pentru start
    hipEventRecord(launch_begin);
    
    // lansam kernel-ul in executie
	sum_matrices<<<num_blocks, block_size>>>(device_a, device_b, device_c, n, n);
    
    // lansam in executie evenimentul pentru stop
    hipEventRecord(launch_end);

    // in loc sa folosim cudaDeviceSynchronize, folosim cudaEventSynchronize
    // prin care se asteapta terminarea thread-urilor
	hipEventSynchronize(launch_end);

    float time = 0;
    // determinam timpul de executie
	hipEventElapsedTime(&time, launch_begin, launch_end);
	
    std::cout << "Time = " << time << std::endl;
    
    hipMemcpy(host_c, &device_c[0], sizeof(float) * n * n, hipMemcpyDeviceToHost);
    for (int i = 0; i < 20; i++) {
        std::cout << host_c[i] << " ";
    }
    std::cout << std::endl;

    // distrugem evenimentele
    hipEventDestroy(launch_begin);
    hipEventDestroy(launch_end);

	hipFree(device_a);
	hipFree(device_b);
    hipFree(device_c);
    
	free(host_a);
	free(host_b);
	free(host_c);

	return 0;
}
