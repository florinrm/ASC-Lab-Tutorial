
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// functie kernel prin care adunam doi arrays
__global__ void vector_add(float *x, float *y, int n) {
    // calculam indexul - echivalent cu for-ul

    // threadId.x - id-ul unui thread blocul actual
	// blockDim.x - dimensiunea blocului actual
	// blockIdx.x - id-ul blocului actual

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) {
        x[i] = x[i] + y[i];
    }
}

int main(void)
{
    const int num_elements = 1 << 16;
    const int num_bytes = num_elements * sizeof(float);

    float *x, *y;

    hipMallocManaged(&x, num_bytes);
    hipMallocManaged(&y, num_bytes);

    if (!x || !y) {
		fprintf(stderr, "[HOST & DEVICE] hipMallocManaged failed\n");
		return 1;
	}

    // se initializeaza x si y
    for (int i = 0; i < num_elements; i++) {
        x[i] = 4;
        y[i] = 2;
    }

    // stabilim dimensiunea unui bloc (adica numarul de threads dintr-un bloc)
    const size_t block_size = 256;
    
    // numarul de blocuri
    size_t blocks_no = num_elements / block_size;
 
    // daca avem un bloc care nu are dimensiunea 256, incrementam numarul de blocuri
    if (num_elements % block_size != 0) {
        ++blocks_no;
    }

    vector_add<<<blocks_no, block_size>>>(x, y, num_elements);
    
    // asteptam ca thread-urile de pe GPU sa-si termine treaba - echivalent cu pthread_join
    hipDeviceSynchronize();


    for (int i = 0; i < 10; ++i) {
        printf("Result %d: %1.1f + %1.1f = %1.3f\n", i, x[i] - y[i], 
                y[i], x[i]);
    }

    // eliberam memoria pe device
    hipFree(x);
    hipFree(y);
  
    return 0;
}