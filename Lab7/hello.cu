
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void kernel_example(int value) {
    printf("[GPU] Hello from the GPU!\n");
    printf("[GPU] The value is %d\n", value);
    printf("[GPU] blockDim = %d, blockId = %d, threadIdx = %d\n", blockDim.x, blockIdx.x, threadIdx.x);
}
 
int main(void) {
    int nDevices;
    printf("[HOST] Hello from the host!\n");
    
    hipGetDeviceCount(&nDevices);
    printf("[HOST] You have %d CUDA-capable GPU(s)\n", nDevices);
    
    // 4 blocuri, fiecare bloc cu 4 threads
    kernel_example<<<4,4>>>(25);
    hipDeviceSynchronize();
    
    return 0;
}