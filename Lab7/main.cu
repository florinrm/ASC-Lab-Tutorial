
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

// functie kernel prin care adunam doi arrays
__global__ void vector_add(float *x, float *y, int n) {
    // calculam indexul - echivalent cu for-ul

    // threadId.x - id-ul unui thread blocul actual
	// blockDim.x - dimensiunea blocului actual
	// blockIdx.x - id-ul blocului actual


    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < n) {
        x[i] = x[i] + y[i];
    }
}

int main(void)
{
    const int num_elements = 1 << 16;
    const int num_bytes = num_elements * sizeof(float);

    float *host_array_x = 0, *host_array_y = 0; // arrays pentru host (CPU)
    float *device_array_x = 0, *device_array_y = 0; // arrays pentru device (GPU)

    // alocam memorie pentru host
    host_array_x = (float *) malloc(num_bytes);
    host_array_y = (float *) malloc(num_bytes);

    // alocam memorie pentru device
    hipMalloc((void **) &device_array_x, num_bytes);
    hipMalloc((void **) &device_array_y, num_bytes);

    // verificam daca alocarea a fost cu succes
    if (host_array_x == 0 || host_array_y == 0 || device_array_x == 0 || device_array_y == 0) {
        printf("[HOST] Couldn't allocate memory\n");
        return 0;
    }

    // se initializeaza x si y
    for (int i = 0; i < num_elements; i++) {
        host_array_x[i] = 4;
        host_array_y[i] = 2;
    }

    // facem transferul host -> device (CPU -> GPU)
    hipMemcpy(device_array_x, host_array_x, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_y, host_array_y, num_bytes, hipMemcpyHostToDevice);

    // stabilim dimensiunea unui bloc (adica numarul de threads dintr-un bloc)
    const size_t block_size = 256;
    
    // numarul de blocuri
    size_t blocks_no = num_elements / block_size;
 
    // daca avem un bloc care nu are dimensiunea 256, incrementam numarul de blocuri
    if (num_elements % block_size != 0) {
        ++blocks_no;
    }

    vector_add<<<blocks_no, block_size>>>(device_array_x, device_array_y, num_elements);
    
    // asteptam ca thread-urile de pe GPU sa-si termine treaba - echivalent cu pthread_join
    // ca apoi sa facem transferul GPU -> CPU
    hipDeviceSynchronize();

    // transferul GPU -> CPU (device -> host)
    hipMemcpy(host_array_x, device_array_x, num_bytes, hipMemcpyDeviceToHost);
    hipMemcpy(host_array_y, device_array_y, num_bytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; ++i) {
        printf("Result %d: %1.1f + %1.1f = %1.3f\n", i, host_array_x[i] - host_array_y[i], 
                host_array_y[i], host_array_x[i]);
    }

    // eliberam memoria pe host
    free(host_array_x);
    free(host_array_y);

    // eliberam memoria pe device
    hipFree(device_array_x);
    hipFree(device_array_y);
  
    return 0;
}